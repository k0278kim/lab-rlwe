#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "params.h"
#include "AES.cuh"
#include "sampler.cuh"
#include "arith_rns.cuh"
#include "randombytes.h"
#include <stdio.h>
#include "consts.cuh"

double std_dev_f(uint64_t *data, uint64_t size) {
    double sum = 0.0, mean, SD = 0, SD2 = 0;
    double max = 1, min = 0;
    uint64_t i;
    max = 1.0;
		for(i=0; i<size; i++)	{
			if(data[i] > max)
				max = data[i];				
		}
		min = max;
		// find the min
		for(i=0; i<size; i++)	{
			if(data[i]< min)
				min = data[i];
		}
    for (i = 0; i < size; ++i) {
        sum += data[i];
    }
    mean = sum / size;
    for (i = 0; i < size; ++i) {
        SD2 += pow(data[i] - mean, 2);
    }
    SD = sqrt(SD2 / size);
    printf("\n max: %.0f \t min: %.0f \tmean: %.4f SD: %.4f\n", max, min, mean, SD);
    return SD;

}


__global__ void gaussian_sampler_S1_gpu(uint8_t *rk, uint32_t *sample)
{	
	uint64_t vx64[4] = {0}, vb_in_64[4] ={0};	
	uint64_t z[8] __attribute__ ((aligned (32)));
	uint64_t b[8] __attribute__ ((aligned (32)));
	uint64_t v64_y1[8] __attribute__ ((aligned (32))) = {0};
	uint64_t v64_y2[8] __attribute__ ((aligned (32))) = {0};
	const uint32_t AES_ROUNDS=3;
	uint32_t i = 8, j = 0, l = 0;
	uint64_t k;//, start_k, stop_k;
	uint8_t *r1;
	uint64_t mod;
	uint32_t mod1, mod2, mod3;
	uint32_t tid = threadIdx.x, bid = blockIdx.x;
	uint8_t r[384] = {0};
	uint32_t rep = 0;// Count no. of AES samp. done in each thread
	while (j < LEN_THREAD)// not adjustable now, one loop 3 samples.
	{
		do
		{			
			if (i == 8)
			{
				for(l=0; l<4; l++) vx64[l] = 0;				
				aes256ctr_squeezeblocks_gpu (r, AES_ROUNDS, (uint32_t*)rk, rep);
				uniform_sampler_S1_gpu(r + 2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE), v64_y1, v64_y2);

				r1 = r;
				cdt_sampler_gpu(r1, vx64);				    			
				for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S1;
				for(l=0; l<4; l++) z[l] = vx64[l] + v64_y1[l];	
				for(l=0; l<4; l++) vb_in_64[l] = z[l] + vx64[l];
				for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y1[l];
				bernoulli_sampler_S1_gpu(b, vb_in_64, r1 + BASE_TABLE_SIZE);

				r1 = r + BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE;
				for(l=0; l<4; l++) vx64[l] = 0;
				cdt_sampler_gpu(r1, vx64);	
				for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S1;
				for(l=0; l<4; l++) z[l+4] = vx64[l] + v64_y2[l];
				for(l=0; l<4; l++) vb_in_64[l] = z[l+4] + vx64[l];
				for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y2[l];
				bernoulli_sampler_S1_gpu(b + 4, vb_in_64, r1 + BASE_TABLE_SIZE);
				i = 0;
				rep++;
			}
			k = (r[2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE) + UNIFORM_REJ * UNIFORM_SIZE] >> i) & 0x1;		
			i++;
		} while (1 ^ ((b[i - 1] & ((z[i - 1] | -z[i - 1]) | (k | -k))) >> 63)); /* rejection condition: b=0 or ((b=1) && (z=0) && (k=0)) */
		mod=z[i-1];

		mod1=mod_prime_gpu(mod, 0);
		mod2=mod_prime_gpu(mod, 1);
		mod3=mod_prime_gpu(mod, 2);		

		sample[j + 0*SIFE_N + bid*SIFE_N*SIFE_NMODULI + tid*LEN_THREAD]=(1-k)*mod1+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[0]-mod1, 0);
		sample[j + 1*SIFE_N + bid*SIFE_N*SIFE_NMODULI + tid*LEN_THREAD]=(1-k)*mod2+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[1]-mod2, 1);
		sample[j + 2*SIFE_N + bid*SIFE_N*SIFE_NMODULI + tid*LEN_THREAD]=(1-k)*mod3+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[2]-mod3, 2);
		j++;
	}
}

__global__ void gaussian_sampler_S2_gpu(uint8_t *rk, uint32_t *sample)
{	
	uint64_t vx64[4] = {0}, vb_in_64[4] ={0};	
	uint64_t z[8] __attribute__ ((aligned (32)));
	uint64_t b[8] __attribute__ ((aligned (32)));
	uint64_t v64_y1[8] __attribute__ ((aligned (32))) = {0};
	uint64_t v64_y2[8] __attribute__ ((aligned (32))) = {0};
	const uint32_t AES_ROUNDS=3;
	uint32_t i = 8, j = 0, l = 0;
	uint64_t k;//, start_k, stop_k;
	uint8_t *r1;
	uint64_t mod;
	uint32_t mod1, mod2, mod3;
	uint32_t tid = threadIdx.x, bid = blockIdx.x;
	uint32_t repeat = blockIdx.y;
	uint8_t r[384] = {0};
	uint32_t rep = 0;// Count no. of AES samp. done in each thread
	while (j < LEN_THREAD)// not adjustable now, one loop 3 samples.
	{
		do
		{			
			if (i == 8)
			{
				for(l=0; l<4; l++) vx64[l] = 0;				
				aes256ctr_squeezeblocks_gpu (r, AES_ROUNDS, (uint32_t*)rk + repeat*4*60, rep);
				uniform_sampler_S2_gpu(r + 2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE), v64_y1, v64_y2);

				r1 = r;
				cdt_sampler_gpu(r1, vx64);				    			
				for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S2;
				for(l=0; l<4; l++) z[l] = vx64[l] + v64_y1[l];	
				for(l=0; l<4; l++) vb_in_64[l] = z[l] + vx64[l];
				for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y1[l];
				bernoulli_sampler_S2_gpu(b, vb_in_64, r1 + BASE_TABLE_SIZE);

				r1 = r + BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE;
				for(l=0; l<4; l++) vx64[l] = 0;
				cdt_sampler_gpu(r1, vx64);	
				for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S2;
				for(l=0; l<4; l++) z[l+4] = vx64[l] + v64_y2[l];
				for(l=0; l<4; l++) vb_in_64[l] = z[l+4] + vx64[l];
				for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y2[l];
				bernoulli_sampler_S2_gpu(b + 4, vb_in_64, r1 + BASE_TABLE_SIZE);
				i = 0;
				rep++;
			}
			k = (r[2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE) + UNIFORM_REJ * UNIFORM_SIZE] >> i) & 0x1;		
			i++;
		} while (1 ^ ((b[i - 1] & ((z[i - 1] | -z[i - 1]) | (k | -k))) >> 63)); /* rejection condition: b=0 or ((b=1) && (z=0) && (k=0)) */
		mod=z[i-1];

		mod1=mod_prime_gpu(mod, 0);
		mod2=mod_prime_gpu(mod, 1);
		mod3=mod_prime_gpu(mod, 2);		

		sample[repeat*SIFE_NMODULI*SIFE_N+ j + 0*SIFE_N + bid*SIFE_N*SIFE_NMODULI+ tid*LEN_THREAD]=(1-k)*mod1+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[0]-mod1, 0);
		sample[repeat*SIFE_NMODULI*SIFE_N+ j + 1*SIFE_N + bid*SIFE_N*SIFE_NMODULI+ tid*LEN_THREAD]=(1-k)*mod2+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[1]-mod2, 1);
		sample[repeat*SIFE_NMODULI*SIFE_N+ j + 2*SIFE_N + bid*SIFE_N*SIFE_NMODULI+ tid*LEN_THREAD]=(1-k)*mod3+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[2]-mod3, 2);
		j++;
	}
}

extern "C" int gaussian_S1_gpu(unsigned char *seed, uint32_t msk[SIFE_L][SIFE_NMODULI][SIFE_N])
{	
	hipEvent_t start, stop;
	uint8_t* dev_rk;
									
	char* m_EncryptKey = (char*)malloc(16 * 15 * sizeof(char));	// Expanded Keys
	uint32_t *d_msk;
	uint64_t *d_clock_c, *clock_c;

	hipEventCreate(&start);	hipEventCreate(&stop);
	hipHostMalloc((void**)&clock_c, SIFE_L*THREAD*sizeof(uint64_t));

	hipMalloc((void**)&dev_rk, 4*60 * sizeof(uint8_t));	//AES256
	hipMalloc((void**)&d_msk, 4*SIFE_L*SIFE_NMODULI*SIFE_N*sizeof(uint32_t));
	hipMalloc((void**)&d_clock_c, SIFE_L*THREAD*sizeof(uint64_t));

	for (int i = 0; i < 15 * 16; i++)	m_EncryptKey[i] = 0;		
	AESPrepareKey(m_EncryptKey, seed, 256);
#ifdef PERF
	hipEventRecord(start);
#endif	
	hipMemcpy(dev_rk, m_EncryptKey, 4*60*sizeof(uint8_t),hipMemcpyHostToDevice);
	gaussian_sampler_S1_gpu<<<SIFE_L, THREAD>>>(dev_rk, d_msk);
#ifdef PERF	
	float elapsed;
	hipEventRecord(stop);
	hipEventSynchronize(stop);  
  hipEventElapsedTime(&elapsed, start, stop);   
  printf("Latency (ms)\n" );
  printf("%.4f \n", elapsed);     
#endif    
  hipMemcpy(msk, d_msk, SIFE_L*SIFE_NMODULI*SIFE_N*sizeof(uint32_t), hipMemcpyDeviceToHost);

	free(m_EncryptKey);
	hipHostFree(clock_c);
	hipFree(dev_rk);
	hipFree(d_msk);
	hipFree(d_clock_c);

	return 0;
}

__device__ uint32_t add_mod_ntt_gpu_2(uint32_t a, uint32_t b, uint32_t sel)
{
	uint64_t c;

	c = (uint64_t)a + (uint64_t)b;

	if (c >= SIFE_MOD_Q_I_gpu[sel]) {
		c -= SIFE_MOD_Q_I_gpu[sel];
	}
	return (uint32_t)c;
}

__global__ void gaussian_sampler_S3_gpu(uint8_t *rk, uint32_t *d_c)
{	
	uint64_t vx64[4] = {0}, vb_in_64[4] ={0};	
	uint64_t z[8] __attribute__ ((aligned (32)));
	uint64_t b[8] __attribute__ ((aligned (32)));
	uint64_t v64_y1[8] __attribute__ ((aligned (32))) = {0};
	uint64_t v64_y2[8] __attribute__ ((aligned (32))) = {0};
	const uint32_t AES_ROUNDS=3;
	uint32_t i = 8, j = 0, l = 0;
	uint64_t k;//, start_k, stop_k;
	uint8_t *r1;
	uint64_t mod;
	uint32_t mod1, mod2, mod3;
	uint32_t tid = threadIdx.x, bid = blockIdx.x;
	uint32_t repeat = blockIdx.y;
	uint8_t r[384] = {0};
	uint32_t rep = 0;// Count no. of AES samp. done in each thread

	uint32_t sample_0, sample_1, sample_2, sample_3;

	if (tid < 512)
	{
		while (j < LEN_THREAD)// not adjustable now, one loop 3 samples.
		{
			do
			{			
				if (i == 8)
				{
					for(l=0; l<4; l++) vx64[l] = 0;				
					aes256ctr_squeezeblocks_gpu (r, AES_ROUNDS, (uint32_t*)rk+ repeat*4*60, rep);
					uniform_sampler_S3_gpu(r + 2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE), v64_y1, v64_y2);

					r1 = r;
					cdt_sampler_gpu(r1, vx64);				    			
					for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S3;
					for(l=0; l<4; l++) z[l] = vx64[l] + v64_y1[l];	
					for(l=0; l<4; l++) vb_in_64[l] = z[l] + vx64[l];
					for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y1[l];
					bernoulli_sampler_S3_gpu(b, vb_in_64, r1 + BASE_TABLE_SIZE);

					r1 = r + BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE;
					for(l=0; l<4; l++) vx64[l] = 0;
					cdt_sampler_gpu(r1, vx64);	
					for(l=0; l<4; l++) vx64[l] = (uint32_t) vx64[l] * BINARY_SAMPLER_K_S3;
					for(l=0; l<4; l++) z[l+4] = vx64[l] + v64_y2[l];
					for(l=0; l<4; l++) vb_in_64[l] = z[l+4] + vx64[l];
					for(l=0; l<4; l++) vb_in_64[l] = (uint32_t) vb_in_64[l] * v64_y2[l];
					bernoulli_sampler_S3_gpu(b + 4, vb_in_64, r1 + BASE_TABLE_SIZE);
					i = 0;
					rep++;
				}
				k = (r[2 * (BASE_TABLE_SIZE + BERNOULLI_TABLE_SIZE) + UNIFORM_REJ * UNIFORM_SIZE] >> i) & 0x1;		
				i++;
			} while (1 ^ ((b[i - 1] & ((z[i - 1] | -z[i - 1]) | (k | -k))) >> 63)); /* rejection condition: b=0 or ((b=1) && (z=0) && (k=0)) */
			mod=z[i-1];

			mod1=mod_prime_gpu(mod, 0);
			mod2=mod_prime_gpu(mod, 1);
			mod3=mod_prime_gpu(mod, 2);		

			sample_0=(1-k)*mod1+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[0]-mod1, 0);
			sample_1=(1-k)*mod2+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[1]-mod2, 1);
			sample_2=(1-k)*mod3+k*mod_prime_gpu(SIFE_MOD_Q_I_GPU[2]-mod3, 2);

			d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+0*SIFE_N+tid*LEN_THREAD] = add_mod_ntt_gpu_2(d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+0*SIFE_N+tid*LEN_THREAD], sample_0, 0);
			d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+1*SIFE_N+tid*LEN_THREAD] = add_mod_ntt_gpu_2(d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+1*SIFE_N+tid*LEN_THREAD], sample_1, 1);
			d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+2*SIFE_N+tid*LEN_THREAD] = add_mod_ntt_gpu_2(d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+2*SIFE_N+tid*LEN_THREAD], sample_2, 2);
			d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+3*SIFE_N+tid*LEN_THREAD] = add_mod_ntt_gpu_2(d_c[repeat*(SIFE_L+1)*SIFE_NMODULI*SIFE_N + bid*SIFE_NMODULI*SIFE_N+j+3*SIFE_N+tid*LEN_THREAD], sample_3, 3);

			j++;
		}
	}
}